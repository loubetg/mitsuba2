#include "hip/hip_runtime.h"
#include <optix_world.h>

#include "random.h"
#include "util.h"

using namespace optix;

rtDeclareVariable(int, fill_surface_interaction, , );

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(void *, accel, , );
rtDeclareVariable(unsigned long long, shape_ptr, , );
rtDeclareVariable(unsigned int, launch_index, rtLaunchIndex, );

rtDeclareVariable(float3, p, attribute p, );
rtDeclareVariable(float2, uv, attribute uv, );
rtDeclareVariable(float3, ns, attribute ns, );
rtDeclareVariable(float3, ng, attribute ng, );
rtDeclareVariable(float3, dp_du, attribute dp_du, );
rtDeclareVariable(float3, dp_dv, attribute dp_dv, );
rtDeclareVariable(Ray, ray, rtCurrentRay,);

rtBuffer<bool> in_mask;

rtBuffer<float> in_ox, in_oy, in_oz,
                in_dx, in_dy, in_dz,
                in_mint, in_maxt, in_kappa;

rtBuffer<float> out_t, out_u, out_v, out_ng_x, out_ng_y,
                out_ng_z, out_ns_x, out_ns_y, out_ns_z,
                out_p_x, out_p_y, out_p_z,
                out_dp_du_x, out_dp_du_y, out_dp_du_z,
                out_dp_dv_x, out_dp_dv_y, out_dp_dv_z;

rtBuffer<unsigned long long> out_shape_ptr;

rtBuffer<uint32_t> out_primitive_id;

rtBuffer<bool> out_hit;

struct PerRayData { };

RT_PROGRAM void ray_gen_closest() {
    float3 ro = make_float3(in_ox[launch_index],
                            in_oy[launch_index],
                            in_oz[launch_index]),
           rd = make_float3(in_dx[launch_index],
                            in_dy[launch_index],
                            in_dz[launch_index]);
    float  mint = in_mint[launch_index],
           maxt = in_maxt[launch_index];

    if (!in_mask[launch_index]) {
        out_shape_ptr[launch_index] = 0;
        out_t[launch_index] = HIP_INF_F;
    } else {
        PerRayData prd;
        Ray ray = make_Ray(ro, rd, 0, mint, maxt);
        rtTrace(top_object, ray, prd);
    }
}

RT_PROGRAM void ray_gen_any() {
    float3 ro = make_float3(in_ox[launch_index],
                            in_oy[launch_index],
                            in_oz[launch_index]),
           rd = make_float3(in_dx[launch_index],
                            in_dy[launch_index],
                            in_dz[launch_index]);
    float  mint = in_mint[launch_index],
           maxt = in_maxt[launch_index];

    Ray ray = make_Ray(ro, rd, 0, mint, maxt);

    if (!in_mask[launch_index]) {
        out_hit[launch_index] = false;
    } else {
        PerRayData prd;
        rtTrace(top_object, ray, prd, RT_VISIBILITY_ALL,
                RT_RAY_FLAG_TERMINATE_ON_FIRST_HIT);
    }
}

RT_PROGRAM void ray_gen_occluder() {
    // if (launch_index == 10)
        // printf("Hello ray_gen_occluder --> launch_index %d, kappa: %f \n", launch_index, in_kappa[launch_index]);

    // TODO should be a variable
    int test_count = 4;

    float kappa = in_kappa[launch_index];

    unsigned int seed = tea<16>(launch_index, 0u);

    // if (launch_index == 0 || launch_index == 262144)
        // printf("--> launch_index %d, kappa: %f, dir %f, seed: %i \n", launch_index, in_kappa[launch_index], in_dx[launch_index], seed);

    float3 ro = make_float3(in_ox[launch_index],
                            in_oy[launch_index],
                            in_oz[launch_index]),
           rd = make_float3(in_dx[launch_index],
                            in_dy[launch_index],
                            in_dz[launch_index]);
    float  mint = in_mint[launch_index],
           maxt = in_maxt[launch_index];

    if (!in_mask[launch_index]) {
        out_shape_ptr[launch_index] = 0;
        out_t[launch_index] = HIP_INF_F;
    } else {
        PerRayData prd;

        float res_t = HIP_INF_F;

        float3 res_p;
        float3 res_ng;
        float2 res_uv;
        unsigned long long res_shape_ptr = 0;
        uint32_t res_prim_id;

        // Compute ray direction frame
        float3 rd_s, rd_t;
        coordinate_system(rd, rd_s, rd_t);

        for (int i = 0; i < test_count; i++) {
            // Sample random direction using vMF
            float2 sample = make_float2(rnd(seed), rnd(seed));
            float3 offset = square_to_von_mises_fisher(sample, kappa);


            // if (launch_index == 10) {
            //     printf("sample: %f, %f \n", sample.x, sample.y);
            //     printf("offset: %f, %f, %f \n", offset.x, offset.y, offset.z);
            //     printf("rd: %f, %f, %f \n", rd.x, rd.y, rd.z);
            //     printf("rd_s: %f, %f, %f \n", rd_s.x, rd_s.y, rd_s.z);
            //     printf("rd_t: %f, %f, %f \n", rd_t.x, rd_t.y, rd_t.z);
            // }

            float3 rd_offset = rd_s * offset.x + rd_t * offset.y + rd * offset.z;

            // Generate and trace ray
            Ray ray = make_Ray(ro, rd_offset, 0, mint, maxt);
            rtTrace(top_object, ray, prd);

            // If no hit, continue
            if (out_t[launch_index] == HIP_INF_F)
                continue;

            // TODO: use ray payload for this
            float3 p1 = make_float3(out_p_x[launch_index], out_p_y[launch_index], out_p_z[launch_index]);
            float3 n1 = make_float3(out_ng_x[launch_index], out_ng_y[launch_index], out_ng_z[launch_index]);
            float2 uv1 = make_float2(out_u[launch_index], out_v[launch_index]);
            unsigned long long shape_ptr1 = out_shape_ptr[launch_index];

            // if (launch_index == 10) {
            //     // printf("launch_index: %d \n", launch_index);
            //     printf("p1: %f, %f, %f \n", p1.x, p1.y, p1.z);
            //     printf("n1: %f, %f, %f \n", n1.x, n1.y, n1.z);
            //     printf("uv1: %f, %f \n", uv1.x, uv1.y);
            //     printf("shape_ptr1: %d \n", shape_ptr1);
            // }

            // -------------------------------------
            // Process intersection


            if (i == 0) {
                res_t = out_t[launch_index];
                res_p = p1;
                res_ng = n1;
                res_uv = uv1;
                res_shape_ptr = shape_ptr1;
                res_prim_id = out_primitive_id[launch_index];
            } else if (res_shape_ptr != shape_ptr1) {
                // Check if ro and p are on the opposite side of the plane defined by {res_o, res_n}
                // If not, then update the res fields with new occluder
                if (dot(n1, res_p - p1) * dot(res_ng, ro - res_p) < 0.f) {
                    res_t = out_t[launch_index];
                    res_p = p1;
                    res_ng = n1;
                    res_uv = uv1;
                    res_shape_ptr = shape_ptr1;
                    res_prim_id = out_primitive_id[launch_index];
                }
            }
        }

        // Write result in ouput SurfaceInteraction3f
        out_t[launch_index] = res_t;

        out_p_x[launch_index] = res_p.x;
        out_p_y[launch_index] = res_p.y;
        out_p_z[launch_index] = res_p.z;

        out_u[launch_index] = res_uv.x;
        out_v[launch_index] = res_uv.y;

        out_shape_ptr[launch_index] = res_shape_ptr;
        out_primitive_id[launch_index] = res_prim_id;
    }
}

RT_PROGRAM void ray_hit() {
    if (out_hit.size() > 0) {
        out_hit[launch_index] = true;
    } else {
        out_shape_ptr[launch_index] = shape_ptr;

        out_primitive_id[launch_index] = rtGetPrimitiveIndex();

        out_u[launch_index] = uv.x;
        out_v[launch_index] = uv.y;

        out_p_x[launch_index] = p.x;
        out_p_y[launch_index] = p.y;
        out_p_z[launch_index] = p.z;

        out_ng_x[launch_index] = ng.x;
        out_ng_y[launch_index] = ng.y;
        out_ng_z[launch_index] = ng.z;

        if (fill_surface_interaction == 1) {
            out_ns_x[launch_index] = ns.x;
            out_ns_y[launch_index] = ns.y;
            out_ns_z[launch_index] = ns.z;

            out_dp_du_x[launch_index] = dp_du.x;
            out_dp_du_y[launch_index] = dp_du.y;
            out_dp_du_z[launch_index] = dp_du.z;

            out_dp_dv_x[launch_index] = dp_dv.x;
            out_dp_dv_y[launch_index] = dp_dv.y;
            out_dp_dv_z[launch_index] = dp_dv.z;
        }

        out_t[launch_index] = sqrt(squared_norm(p - ray.origin) / squared_norm(ray.direction));
    }
}

RT_PROGRAM void ray_miss() {
    if (out_hit.size() > 0) {
        out_hit[launch_index] = false;
    } else {
        out_shape_ptr[launch_index] = 0;
        out_t[launch_index] = HIP_INF_F;
    }
}

RT_PROGRAM void ray_err() {
    rtPrintExceptionDetails();
}
